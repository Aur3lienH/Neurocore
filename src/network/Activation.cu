#include "hip/hip_runtime.h"
#include "network/Activation.cuh"
#include "network/InitFunc.cuh"
#include <fstream>
#include <emmintrin.h>
#include <cmath>


Activation::Activation()
{
    ID = -1;
}

std::string Activation::getName() const
{
    return name;
}

MAT* Activation::InitBiases(const int outputSize)
{
    return new MAT(outputSize, 1, 1);
}

#if USE_GPU

void Activation::FeedForward(const MAT* input, const hipdnnTensorDescriptor_t& inputDesc, MAT* output,
                             const hipdnnTensorDescriptor_t& outputDesc)
#else

void Activation::FeedForward(const MAT* input, MAT* output)
#endif
{
#if SAFE
    if (input->GetCols() != output->GetCols() || input->GetRows() != output->GetRows() ||
        input->GetDims() != output->GetDims())
    {
        throw std::invalid_argument("Activation::FeedForward : Both matrix must have the same shape !");
    }
#endif

#if USE_GPU
    checkCUDNN(hipdnnActivationForward(Matrix_GPU::cuda->cudnnHandle, activationDesc, &Matrix_GPU::cuda->one,
                                      inputDesc, input->GetData(), &Matrix_GPU::cuda->zero,
                                      outputDesc, output->GetData()));
#else
    for (int i = 0; i < input->GetSize(); i++)
    {
        output[0][i] = Function(input[0][i]);
    }
#endif
}


#if USE_GPU

void Activation::Derivative(const MAT* input, const hipdnnTensorDescriptor_t& inputDesc, const MAT* lastDelta,
                            const hipdnnTensorDescriptor_t& lastDeltaDesc, const MAT* z,
                            const hipdnnTensorDescriptor_t& zDesc,
                            MAT* output, const hipdnnTensorDescriptor_t& outputDesc)
#else

void Activation::Derivative(const MAT* input, MAT* output)
#endif
{
#if USE_GPU
    checkCUDNN(hipdnnActivationBackward(Matrix_GPU::cuda->cudnnHandle, activationDesc, &Matrix_GPU::cuda->one,
                                       inputDesc, input->GetData(),
                                       lastDeltaDesc,
                                       lastDelta->GetData(), zDesc, z->GetData(),
                                       &Matrix_GPU::cuda->zero, outputDesc, output->GetData()));

#else

    if (input->GetCols() != output->GetCols() || input->GetRows() != output->GetRows() ||
        input->GetDims() != output->GetDims())
    {
        throw std::invalid_argument("Activation::Derivative() : Both matrix must have the same shape !");
    }

    for (int i = 0; i < input->GetSize(); i++)
    {
        output[0][i] = Derive(input[0][i]);
    }
#endif
}

void Activation::Save(std::ofstream& writer)
{
    writer.write(reinterpret_cast<char*>(&ID), sizeof(int));
}

Activation* Activation::Read(std::ifstream& reader)
{
    int ID;
    reader.read(reinterpret_cast<char*>(&ID), sizeof(int));
    if (ID == 0)
    {
        return new Sigmoid();
    }
    else if (ID == 1)
    {
        return new SigmoidPrime();
    }
    else if (ID == 2)
    {
        return new ReLU();
    }
    else if (ID == 3)
    {
        float f;
        reader.read(reinterpret_cast<char*>(&f), sizeof(float));
        return new LeakyReLU(f);
    }
    else if (ID == 4)
    {
        return new Softmax();
    }
    else if (ID == 5)
    {
        return new Tanh();
    }
    else
    {
        throw std::invalid_argument("Invalid ID for loading activation function");
    }
}

#if USE_GPU

void Activation::Function(const MAT& input, const hipdnnTensorDescriptor_t& inputDesc, MAT& output,
                          const hipdnnTensorDescriptor_t& outputDesc)
{
    checkCUDNN(hipdnnActivationForward(Matrix_GPU::cuda->cudnnHandle, activationDesc, &Matrix_GPU::cuda->one,
                                      inputDesc, input.GetData(), &Matrix_GPU::cuda->zero,
                                      outputDesc, output.GetData()));
}

#endif

Sigmoid::Sigmoid()
{
    name = "Sigmoid";
    ID = 0;
#if USE_GPU
    checkCUDNN(hipdnnCreateActivationDescriptor(&activationDesc));
    checkCUDNN(
            hipdnnSetActivationDescriptor(activationDesc, HIPDNN_ACTIVATION_SIGMOID, HIPDNN_NOT_PROPAGATE_NAN, 0));
#endif
}

#if not USE_GPU

double Sigmoid::Function(const double input)
{
    return 1 / (1 + exp(-input));
}

#endif

double Sigmoid::Derive(const double input)
{
    return exp(-input) / pow(1 + exp(-input), 2);
}

MAT* Sigmoid::InitWeights(const int previousNeuronsCount, const int NeuronsCount)
{
#if USE_GPU
    auto* weights = new Matrix_GPU(NeuronsCount, previousNeuronsCount);
#else
    auto* weights = new MAT(NeuronsCount, previousNeuronsCount, 1, true);
#endif
    WeightsInit::XavierInit(previousNeuronsCount, weights);
    return weights;
}

SigmoidPrime::SigmoidPrime()
{
    name = "SigmoidPrime";
    ID = 1;
#if USE_GPU
    throw std::runtime_error("The sigmoid prime class has no meaning on GPU, please use the sigmoid class instead");
#endif
}

#if not USE_GPU

double SigmoidPrime::Function(const double input)
{
    return 0.5 + 0.5 * tanh(0.5 * input);
}

#endif

double SigmoidPrime::Derive(const double input)
{
    return 0.5 * (1 + tanh(0.5 * input)) * (1 - tanh(0.5 * input));
}

MAT* SigmoidPrime::InitWeights(const int previousNeuronsCount, const int NeuronsCount)
{
#if USE_GPU
    auto* weights = new Matrix_GPU(NeuronsCount, previousNeuronsCount);
#else
    auto* weights = new Matrix(NeuronsCount, previousNeuronsCount, 1, true);
#endif
    WeightsInit::XavierInit(previousNeuronsCount, weights);
    return weights;
}

ReLU::ReLU()
{
    name = "ReLU";
    ID = 2;
#if USE_GPU
    checkCUDNN(hipdnnCreateActivationDescriptor(&activationDesc));
    checkCUDNN(
            hipdnnSetActivationDescriptor(activationDesc, HIPDNN_ACTIVATION_RELU, HIPDNN_NOT_PROPAGATE_NAN, 0));
#endif
}

#if not USE_GPU

void ReLU::FeedForward(const MAT* input, MAT* output)
{
    __m128 zero = _mm_setzero_ps();

    size_t i;
    for (i = 0; i <= input->GetSize() - 4; i += 4)
    {
        __m128 vals = _mm_loadu_ps(&((*input)[i]));
        __m128 result = _mm_max_ps(zero, vals);
        _mm_storeu_ps(&((*output)[i]), result);
    }

    // Process any remaining values
    for (; i < input->GetSize(); ++i)
    {
        if ((*input)[i] < 0) (*output)[i] = 0;
    }
}

#endif

#if not USE_GPU

void ReLU::Derivative(const MAT* input, MAT* output)
{
    __m128 zero = _mm_setzero_ps();
    __m128 one = _mm_set1_ps(1.0);

    int i;
    for (i = 0; i <= input->GetSize() - 4; i += 4)
    {
        __m128 vals = _mm_loadu_ps(&((*input)[i]));
        __m128 mask = _mm_cmpgt_ps(vals,
                                   zero); // Create a mask where each element is either 0xFFFFFFFFFFFFFFFF if vals > 0 or 0x0 otherwise
        __m128 result = _mm_and_ps(one, mask);  // Set to 1.0 where mask is true
        _mm_storeu_ps(&((*output)[i]), result);
    }

    // Process any remaining values
    for (; i < input->GetSize(); ++i)
    {
        (*output)[i] = ((*input)[i] > 0) ? 1.0 : 0.0;
    }
}

double ReLU::Function(const double input)
{
    if (input > 0)
    {
        return input;
    }
    else
    {
        return 0;
    }
}

#endif

double ReLU::Derive(const double input)
{
    if (input > 0)
    {
        return 1;
    }
    else
    {
        return 0;
    }
}

MAT* ReLU::InitWeights(const int previousNeuronsCount, const int NeuronsCount)
{
#if USE_GPU
    auto* weights = new Matrix_GPU(NeuronsCount, previousNeuronsCount);
#else
    auto* weights = new Matrix(NeuronsCount, previousNeuronsCount, 1, true);
#endif
    WeightsInit::HeUniform(previousNeuronsCount, weights);
    return weights;
}

MAT* ReLU::InitBiases(const int outputSize)
{
#if USE_GPU
    float* biases = new float[outputSize];
    for (int i = 0; i < outputSize; i++)
        biases[i] = 0.01f;

    Matrix_GPU* res = new Matrix_GPU(outputSize, 1);
    checkCUDA(hipMemcpy(res->GetData(), biases, outputSize * sizeof(float), hipMemcpyHostToDevice));
    delete[] biases;

    return res;
#else
    return new MAT(outputSize, 1, 0.01f);
#endif
}

LeakyReLU::LeakyReLU(const double _alpha)
{
    alpha = _alpha;
    ID = 3;
#if USE_GPU
    throw std::runtime_error("LeakyReLU is not implemented on GPU");
#endif
}

#if not USE_GPU

double LeakyReLU::Function(const double input)
{
    return input > 0 ? input : 0.01 * input;
}

#endif

double LeakyReLU::Derive(const double input)
{
    return input > 0 ? 1 : 0.01;
}

void LeakyReLU::Save(std::ofstream& writer)
{
    writer.write(reinterpret_cast<char*>(&ID), sizeof(int));
    writer.write(reinterpret_cast<char*>(&alpha), sizeof(float));
}

MAT* LeakyReLU::InitWeights(const int previousNeuronsCount, const int NeuronsCount)
{
#if USE_GPU
    auto* weights = new Matrix_GPU(NeuronsCount, previousNeuronsCount);
#else
    auto* weights = new Matrix(NeuronsCount, previousNeuronsCount, 1, true);
#endif
    WeightsInit::HeUniform(previousNeuronsCount, weights);
    return weights;
}

Softmax::Softmax()
{
    name = "Softmax";
    ID = 4;
}

#if USE_GPU

void Softmax::FeedForward(const MAT* input, const hipdnnTensorDescriptor_t& inputDesc, MAT* output,
                          const hipdnnTensorDescriptor_t& outputDesc)
#else

void Softmax::FeedForward(const MAT* input, MAT* output)
#endif
{
#if USE_GPU
    checkCUDNN(hipdnnSoftmaxForward(Matrix_GPU::cuda->cudnnHandle, HIPDNN_SOFTMAX_ACCURATE, HIPDNN_SOFTMAX_MODE_INSTANCE,
                                   &Matrix_GPU::cuda->one, inputDesc, input->GetData(),
                                   &Matrix_GPU::cuda->zero, outputDesc, output->GetData()));
#else
    double sum = 0;
    double max = input[0][0];
    for (int i = 0; i < input->GetSize(); i++)
    {
        if (input[0][i] > max)
        {
            max = input[0][i];
        }
    }

    for (int i = 0; i < input->GetSize(); i++)
    {
        sum += exp(input[0][i] - max);
    }
    for (int i = 0; i < input->GetSize(); i++)
    {
        output[0][i] = exp(input[0][i] - max) / sum;
    }
#endif
}

MAT* Softmax::InitWeights(const int previousNeuronsCount, const int NeuronsCount)
{
    MAT* weights = new MAT(NeuronsCount, previousNeuronsCount);
    WeightsInit::XavierInit(previousNeuronsCount, weights);
    return weights;
}

#if USE_GPU

void Softmax::Derivative(const MAT* input, const hipdnnTensorDescriptor_t& inputDesc, const MAT* lastDelta,
                         const hipdnnTensorDescriptor_t& lastDeltaDesc, const MAT* z,
                         const hipdnnTensorDescriptor_t& zDesc,
                         MAT* output, const hipdnnTensorDescriptor_t& outputDesc)
{
    /*checkCUDNN(hipdnnSoftmaxBackward(Matrix_GPU::cuda->cudnnHandle, HIPDNN_SOFTMAX_ACCURATE, HIPDNN_SOFTMAX_MODE_INSTANCE,
                                    &Matrix_GPU::cuda->one, *input->GetDescriptor_1D(), input->GetData(),
                                    *lastDelta->GetDescriptor_1D(), lastDelta->GetData(), &Matrix_GPU::cuda->zero,
                                    *output->GetDescriptor_1D(), output->GetData()));*/

    // The CPU version sets all values of output to one, but as the GPU version of Derivative also multiplies output
    // by lastDelta, we can just copy lastDelta to output
    checkCUDA(hipMemcpy(output->GetData(), lastDelta->GetData(), output->GetSize() * sizeof(float),
                         hipMemcpyHostToDevice));
}

#else

void Softmax::Derivative(const MAT* input, MAT* output)
{
    for (int i = 0; i < input->GetSize(); i++)
    {
        output[0][i] = 1;
    }
}

#endif


Tanh::Tanh()
{
    name = "Tanh";
    ID = 5;
#if USE_GPU
    checkCUDNN(hipdnnCreateActivationDescriptor(&activationDesc));
    checkCUDNN(hipdnnSetActivationDescriptor(activationDesc, HIPDNN_ACTIVATION_TANH, HIPDNN_NOT_PROPAGATE_NAN, 0));
#endif
}

#if not USE_GPU

double Tanh::Function(const double input)
{
    return tanh(input);
}

#endif

double Tanh::Derive(const double input)
{
    return 1 - tanh(input) * tanh(input);
}

MAT* Tanh::InitWeights(const int previousNeuronsCount, const int NeuronsCount)
{
#if USE_GPU
    auto* weights = new Matrix_GPU(NeuronsCount, previousNeuronsCount);
#else
    auto* weights = new Matrix(NeuronsCount, previousNeuronsCount, 1, true);
#endif
    WeightsInit::XavierInit(previousNeuronsCount, weights);
    return weights;
}

/*
None::None() : Activation()
{

}

#if not USE_GPU

double None::Function(const double input)
{
    return 0;
}

#endif
*/







