#include "network/layers/MaxPooling.cuh"


MaxPoolLayer::MaxPoolLayer(int filterSize, int stride) : PoolingLayer(filterSize, stride)
{
    LayerID = 4;
}

const MAT* MaxPoolLayer::FeedForward(const MAT* input)
{
#if USE_GPU
    checkCUDNN(
            hipdnnPoolingForward(Matrix_GPU::cuda->cudnnHandle,
                                poolingDescriptor,
                                &Matrix_GPU::cuda->one,
                                forwardInputDesc,
                                input->GetData(),
                                &Matrix_GPU::cuda->zero,
                                forwardOutputDesc,
                                result->GetData()));
#else
    result->Reshape(layerShape->dimensions[0], layerShape->dimensions[1], layerShape->dimensions[2]);
    Matrix::MaxPool(input, result, filterSize, stride);
#endif

    return result;
}

MAT* MaxPoolLayer::BackPropagate(const MAT* delta, const MAT* previousActivation)
{
#if USE_GPU
    checkCUDNN(
            hipdnnPoolingBackward(Matrix_GPU::cuda->cudnnHandle,
                                 poolingDescriptor,
                                 &Matrix_GPU::cuda->one,
                                 forwardOutputDesc,
                                 result->GetData(),
                                 forwardOutputDesc,
                                 delta->GetData(),
                                 forwardInputDesc,
                                 previousActivation->GetData(),
                                 &Matrix_GPU::cuda->zero,
                                 forwardInputDesc,
                                 newDelta->GetData()));
#else
    // The idea is that if an element is the maximum than maxPool has selected, then the delta is
    // the same as the previous delta, because the current element is the only one affecting the result.

    for (int m = 0; m < layerShape->dimensions[2]; m++)
    {
        for (int i = 0; i < layerShape->dimensions[0]; ++i)
        {
            for (int j = 0; j < layerShape->dimensions[1]; ++j)
            {
                for (int k = 0; k < filterSize; ++k)
                {
                    for (int l = 0; l < filterSize; ++l)
                    {
                        const int r = i * stride + k;
                        //if (r >= previousActivation->GetRows())
                        //    continue;
                        const int c = j * stride + l;
                        //if (c >= previousActivation->GetCols())
                        //    continue;
                        //std::cout << m  << "  " << i << "  " << j << "  " << k << "  " << l << "\n";
                        //std::cout << r << " : x y : " << c << "\n";
                        //std::cout << (*previousActivation)(r,c) << "\n";

                        if (r >= previousActivation->GetRows())
                            continue;
                        if (c >= previousActivation->GetCols())
                            continue;


                        if ((*previousActivation)(r, c) == (*result)(i, j))
                            (*newDelta)(r, c) = (*delta)(i, j);
                        // Should already be 0
                        //else
                        //    (*newDelta)(r,c) = 0.0;
                    }
                }
            }
        }
        previousActivation->GoToNextMatrix();
        result->GoToNextMatrix();
        newDelta->GoToNextMatrix();
        delta->GoToNextMatrix();
    }


    previousActivation->ResetOffset();
    result->ResetOffset();
    newDelta->ResetOffset();
    delta->ResetOffset();
#endif


    //std::cout << *delta;

    return newDelta;
}

std::string MaxPoolLayer::getLayerTitle()
{
    std::string buf;
    buf += "MaxPool Layer\n";
    buf += "Size: " + std::to_string(filterSize) + "\n";
    buf += "Stride: " + std::to_string(stride) + "\n";
    buf += "Output : " + layerShape->GetDimensions() + "\n";
    return buf;
}

Layer* MaxPoolLayer::Clone()
{
    return new MaxPoolLayer(filterSize, stride);
}

Layer* MaxPoolLayer::Load(std::ifstream& reader)
{
    int _filterSize;
    int _tempStride;
    reader.read(reinterpret_cast<char*>(&_filterSize), sizeof(int));
    reader.read(reinterpret_cast<char*>(&_tempStride), sizeof(int));
    return new MaxPoolLayer(_filterSize, _tempStride);
}

void MaxPoolLayer::SpecificSave(std::ofstream& writer)
{
    int tempFilterSize = filterSize;
    int tempStride = stride;
    writer.write(reinterpret_cast<char*>(&tempFilterSize), sizeof(int));
    writer.write(reinterpret_cast<char*>(&tempStride), sizeof(int));
}

#if USE_GPU

void MaxPoolLayer::Compile(LayerShape* previousActivation)
{
    PoolingLayer::Compile(previousActivation);
    checkCUDNN(hipdnnCreatePoolingDescriptor(&poolingDescriptor));
    checkCUDNN(hipdnnSetPooling2dDescriptor(poolingDescriptor,
                                           HIPDNN_POOLING_MAX,
                                           HIPDNN_NOT_PROPAGATE_NAN,
                                           filterSize,
                                           filterSize,
                                           0,
                                           0,
                                           stride,
                                           stride));
}

#endif
