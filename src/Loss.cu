#include "hip/hip_runtime.h"
#include "Loss.cuh"
#include <cmath>
#include "Matrix.cuh"


Loss::Loss()
{
}

void Loss::Save(std::ofstream& writer)
{
    writer.write(reinterpret_cast<char*>(&ID), sizeof(int));
}

Loss* Loss::Read(std::ifstream& reader)
{
    int id;
    reader.read(reinterpret_cast<char*>(&id), sizeof(int));
    if (id == 0)
    {
        return new MSE();
    }
    else if (id == 1)
    {
        return new CrossEntropy();
    }
    else
    {
        throw std::invalid_argument("Invalid ID : Loss function");
    }
}


MSE::MSE()
{
    ID = 0;
}

double MSE::Cost(const MAT* output, const MAT* target)
{
    double cost = 0;
#if USE_GPU
    std::cout << "MSE::Cost kernel not implemented yet\n";
    Matrix outputCPU(output->GetRows(), output->GetCols(), output->GetDims());
    checkCUDA(hipMemcpy(outputCPU.GetData(), output->GetData(), output->GetSize() * sizeof(float),
                         hipMemcpyDeviceToHost));
    Matrix targetCPU(target->GetRows(), target->GetCols(), target->GetDims());
    checkCUDA(hipMemcpy(targetCPU.GetData(), target->GetData(), target->GetSize() * sizeof(float),
                         hipMemcpyDeviceToHost));
#endif
    for (int i = 0; i < output->GetRows() * output->GetCols(); i++)
    {
#if USE_GPU
        cost += pow(outputCPU[i] - targetCPU[i], 2);
#else
        cost += pow(output[0][i] - target[0][i], 2);
#endif
    }
    return cost / (2 * output->GetRows());
}

void MSE::CostDerivative(const MAT* output, const MAT* target, MAT* result)
{
#if USE_GPU
    std::cout << "MSE::CostDerivative kernel not implemented yet\n";
    Matrix outputCPU(output->GetRows(), output->GetCols(), output->GetDims());
    checkCUDA(hipMemcpy(outputCPU.GetData(), output->GetData(), output->GetSize() * sizeof(float),
                         hipMemcpyDeviceToHost));
    Matrix targetCPU(target->GetRows(), target->GetCols(), target->GetDims());
    checkCUDA(hipMemcpy(targetCPU.GetData(), target->GetData(), target->GetSize() * sizeof(float),
                         hipMemcpyDeviceToHost));
    Matrix resultCPU(result->GetRows(), result->GetCols(), result->GetDims());
    checkCUDA(hipMemcpy(resultCPU.GetData(), result->GetData(), result->GetSize() * sizeof(float),
                         hipMemcpyDeviceToHost));
#endif
    for (int i = 0; i < output->GetRows() * output->GetCols(); i++)
    {
#if USE_GPU
        resultCPU[i] = outputCPU[i] - targetCPU[i];
#else
        result[0][i] = output[0][i] - target[0][i];
#endif
    }

#if USE_GPU
    checkCUDA(hipMemcpy(result->GetData(), resultCPU.GetData(), result->GetSize() * sizeof(float),
                         hipMemcpyHostToDevice));
#endif
}


CrossEntropy::CrossEntropy()
{
    ID = 1;
}

double CrossEntropy::Cost(const MAT* output, const MAT* target)
{
    double cost = 0;
#if USE_GPU
    std::cout << "CrossEntropy::Cost kernel not implemented yet\n";
    Matrix outputCPU(output->GetRows(), output->GetCols(), output->GetDims());
    checkCUDA(hipMemcpy(outputCPU.GetData(), output->GetData(), output->GetSize() * sizeof(float),
                         hipMemcpyDeviceToHost));
    Matrix targetCPU(target->GetRows(), target->GetCols(), target->GetDims());
    checkCUDA(hipMemcpy(targetCPU.GetData(), target->GetData(), target->GetSize() * sizeof(float),
                         hipMemcpyDeviceToHost));
#endif
    for (int i = 0; i < output->GetRows() * output->GetCols(); i++)
    {
#if USE_GPU
        cost += targetCPU[i] * log(outputCPU[i] + 1e-15) + (1 - targetCPU[i]) * log(1 - outputCPU[i] + 1e-15);
#else
        cost += target[0][i] * log(output[0][i] + 1e-15) + (1 - target[0][i]) * log(1 - output[0][i] + 1e-15);
#endif
    }

#if USE_GPU
    checkCUDA(hipMemcpy(output->GetData(), outputCPU.GetData(), output->GetSize() * sizeof(float),
                         hipMemcpyHostToDevice));
    checkCUDA(hipMemcpy(target->GetData(), targetCPU.GetData(), target->GetSize() * sizeof(float),
                         hipMemcpyHostToDevice));
#endif

    return -cost / output->GetRows();
}

__global__
void CostDerivativeKernel(const float* output, const float* target, float* result, const int size)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size)
    {
        if (target[i] == 1)
        {
            result[i] = -1 + output[i];
        }
        else
        {
            result[i] = output[i];
        }
    }
}

void CrossEntropy::CostDerivative(const MAT* output, const MAT* target, MAT* result)
{
#if USE_GPU
    const int blocksPerGrid =
            (output->GetSize() + Matrix_GPU::cuda->threadsPerBlock - 1) / Matrix_GPU::cuda->threadsPerBlock;
    CostDerivativeKernel<<<blocksPerGrid, Matrix_GPU::cuda->threadsPerBlock>>>(output->GetData(), target->GetData(),
                                                                               result->GetData(), output->GetSize());
    checkCUDA(hipDeviceSynchronize());
#else
    for (int i = 0; i < output->GetRows() * output->GetCols(); i++)
    {
        if (target[0][i] == 1)
        {
            result[0][i] = -1 + output[0][i];
        }
        else
        {
            result[0][i] = output[0][i];
        }
    }
#endif
}





