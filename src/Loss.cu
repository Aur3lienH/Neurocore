#include "hip/hip_runtime.h"
#include "Loss.cuh"
#include <cmath>
#include "Matrix.cuh"


Loss::Loss()
{
}

void Loss::Save(std::ofstream& writer)
{
    writer.write(reinterpret_cast<char*>(&ID), sizeof(int));
}

Loss* Loss::Read(std::ifstream& reader)
{
    int id;
    reader.read(reinterpret_cast<char*>(&id), sizeof(int));
    if (id == 0)
    {
        return new MSE();
    }
    else if (id == 1)
    {
        return new CrossEntropy();
    }
    else
    {
        throw std::invalid_argument("Invalid ID : Loss function");
    }
}


MSE::MSE()
{
    ID = 0;
}

double MSE::Cost(const MAT* output, const MAT* target)
{
    double cost = 0;
#if USE_GPU
    std::cout << "MSE::Cost kernel not implemented yet\n";
    Matrix outputCPU(output->GetRows(), output->GetCols(), output->GetDims());
    checkCUDA(hipMemcpy(outputCPU.GetData(), output->GetData(), output->GetSize() * sizeof(float),
                         hipMemcpyDeviceToHost));
    Matrix targetCPU(target->GetRows(), target->GetCols(), target->GetDims());
    checkCUDA(hipMemcpy(targetCPU.GetData(), target->GetData(), target->GetSize() * sizeof(float),
                         hipMemcpyDeviceToHost));
#endif
    for (int i = 0; i < output->GetRows() * output->GetCols(); i++)
    {
#if USE_GPU
        cost += pow(outputCPU[i] - targetCPU[i], 2);
#else
        cost += pow(output[0][i] - target[0][i], 2);
#endif
    }
    return cost / (2 * output->GetRows());
}

void MSE::CostDerivative(const MAT* output, const MAT* target, MAT* result)
{
#if USE_GPU
    std::cout << "MSE::CostDerivative kernel not implemented yet\n";
    Matrix outputCPU(output->GetRows(), output->GetCols(), output->GetDims());
    checkCUDA(hipMemcpy(outputCPU.GetData(), output->GetData(), output->GetSize() * sizeof(float),
                         hipMemcpyDeviceToHost));
    Matrix targetCPU(target->GetRows(), target->GetCols(), target->GetDims());
    checkCUDA(hipMemcpy(targetCPU.GetData(), target->GetData(), target->GetSize() * sizeof(float),
                         hipMemcpyDeviceToHost));
    Matrix resultCPU(result->GetRows(), result->GetCols(), result->GetDims());
    checkCUDA(hipMemcpy(resultCPU.GetData(), result->GetData(), result->GetSize() * sizeof(float),
                         hipMemcpyDeviceToHost));
#endif
    for (int i = 0; i < output->GetRows() * output->GetCols(); i++)
    {
#if USE_GPU
        resultCPU[i] = outputCPU[i] - targetCPU[i];
#else
        result[0][i] = output[0][i] - target[0][i];
#endif
    }

#if USE_GPU
    checkCUDA(hipMemcpy(result->GetData(), resultCPU.GetData(), result->GetSize() * sizeof(float),
                         hipMemcpyHostToDevice));
#endif
}


CrossEntropy::CrossEntropy()
{
    ID = 1;
}

double CrossEntropy::Cost(const MAT* output, const MAT* target)
{
    double cost = 0;
#if USE_GPU
    std::cout << "CrossEntropy::Cost kernel not implemented yet\n";
    Matrix outputCPU(output->GetRows(), output->GetCols(), output->GetDims());
    checkCUDA(hipMemcpy(outputCPU.GetData(), output->GetData(), output->GetSize() * sizeof(float),
                         hipMemcpyDeviceToHost));
    Matrix targetCPU(target->GetRows(), target->GetCols(), target->GetDims());
    checkCUDA(hipMemcpy(targetCPU.GetData(), target->GetData(), target->GetSize() * sizeof(float),
                         hipMemcpyDeviceToHost));
#endif
    for (int i = 0; i < output->GetRows() * output->GetCols(); i++)
    {
#if USE_GPU
        cost += targetCPU[i] * log(outputCPU[i] + 1e-15) + (1 - targetCPU[i]) * log(1 - outputCPU[i] + 1e-15);
#else
        cost += target[0][i] * log(output[0][i] + 1e-15) + (1 - target[0][i]) * log(1 - output[0][i] + 1e-15);
#endif
    }
    return -cost / output->GetRows();
}

void CrossEntropy::CostDerivative(const MAT* output, const MAT* target, MAT* result)
{
#if USE_GPU
    std::cout << "CrossEntropy::CostDerivative kernel not implemented yet\n";
    Matrix outputCPU(output->GetRows(), output->GetCols(), output->GetDims());
    checkCUDA(hipMemcpy(outputCPU.GetData(), output->GetData(), output->GetSize() * sizeof(float),
                         hipMemcpyDeviceToHost));
    Matrix targetCPU(target->GetRows(), target->GetCols(), target->GetDims());
    checkCUDA(hipMemcpy(targetCPU.GetData(), target->GetData(), target->GetSize() * sizeof(float),
                         hipMemcpyDeviceToHost));
    Matrix resultCPU(result->GetRows(), result->GetCols(), result->GetDims());
    checkCUDA(hipMemcpy(resultCPU.GetData(), result->GetData(), result->GetSize() * sizeof(float),
                         hipMemcpyDeviceToHost));
#endif
    for (int i = 0; i < output->GetRows() * output->GetCols(); i++)
    {
#if USE_GPU
        if (targetCPU[i] == 1)
        {
            resultCPU[i] = -1 / outputCPU[i];
        }
        else
        {
            resultCPU[i] = outputCPU[i];
        }
#else
        if (target[0][i] == 1)
        {
            result[0][i] = -1 + output[0][i];
        }
        else
        {
            result[0][i] = output[0][i];
        }
#endif
    }

#if USE_GPU
    checkCUDA(hipMemcpy(result->GetData(), resultCPU.GetData(), result->GetSize() * sizeof(float),
                         hipMemcpyHostToDevice));
#endif
}





