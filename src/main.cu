#include "hip/hip_runtime.h"
#include "Examples/Mnist.cuh"
#include "Quickdraw.cuh"

int main()
{
    //Mnist2();
    QuickDraw2(10000);

#if USE_GPU
    delete Matrix_GPU::cuda;
#endif
    return 0;
    //LoadAndTest("./Models/MNIST_11.net",true);
}


/*#include "hipDNN.h"
#include <iostream>
#include "Matrix.cuh"

#define checkCUDNN(expression)                               \
  {                                                          \
    hipdnnStatus_t status = (expression);                     \
    if (status != HIPDNN_STATUS_SUCCESS) {                    \
      std::cerr << "Error on line " << __LINE__ << ": "      \
                << hipdnnGetErrorString(status) << std::endl; \
      std::exit(EXIT_FAILURE);                               \
    }                                                        \
  }

#include <hipDNN.h>
#include <iostream>

using namespace std;


int main()
{
    hipdnnHandle_t cudnn;
    hipdnnTensorDescriptor_t input_desc;
    hipdnnTensorDescriptor_t output_desc;
    hipdnnFilterDescriptor_t filter_desc;
    hipdnnConvolutionDescriptor_t conv_desc;
    hipdnnTensorDescriptor_t bias_desc;

    hipdnnConvolutionFwdAlgo_t falgo;
    hipdnnConvolutionBwdFilterAlgo_t b_falgo;
    hipdnnConvolutionBwdDataAlgo_t b_dalgo;

    float* d_input = nullptr;
    float* d_output = nullptr;
    float* d_filter = nullptr;
    float* d_bias = nullptr;

    int input_n = 1;
    int input_c = 1;
    int input_h = 5;
    int input_w = 5;

    // output size
    int output_n = input_n;
    int output_c = 2;
    int output_h = 1;
    int output_w = 1;

    // kernel size
    int filter_h = 3;
    int filter_w = 3;

    // alpha, beta
    float one = 1.f;
    float zero = 0.f;

    std::cout << "[" << __LINE__ << "]" << std::endl;

    hipdnnCreate(&cudnn);

    std::cout << "[" << __LINE__ << "]" << std::endl;

    // Create Resources
    hipdnnCreateTensorDescriptor(&input_desc);
    hipdnnCreateTensorDescriptor(&output_desc);
    hipdnnCreateFilterDescriptor(&filter_desc);
    hipdnnCreateConvolutionDescriptor(&conv_desc);
    hipdnnCreateTensorDescriptor(&bias_desc);

    std::cout << "[" << __LINE__ << "]" << std::endl;

    // Initilziae resources
    hipdnnSetTensor4dDescriptor(input_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, input_n, input_c, input_h, input_w);
    hipdnnSetFilter4dDescriptor(filter_desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, output_c, input_c, filter_h, filter_w);
    hipdnnSetConvolution2dDescriptor(conv_desc,
                                    0, 0,
                                    1, 1,
                                    1, 1,
                                    HIPDNN_CROSS_CORRELATION,
                                    HIPDNN_DATA_FLOAT);
    hipdnnGetConvolution2dForwardOutputDim(conv_desc, input_desc, filter_desc, &output_n, &output_c, &output_h,
                                          &output_w);
    hipdnnSetTensor4dDescriptor(output_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, output_n, output_c, output_h,
                               output_w);
    hipdnnSetTensor4dDescriptor(bias_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, output_c, 1, 1);

    int weight_size = output_c * input_c * filter_h * filter_w;
    int bias_size = output_c;

    std::cout << "input  size: " << input_n << " " << input_c << " " << input_h << " " << input_w << std::endl;
    std::cout << "output size: " << output_n << " " << output_c << " " << output_h << " " << output_w << std::endl;

    std::cout << "[" << __LINE__ << "]" << std::endl;

    // convolution
    size_t workspace_size = 0;
    size_t temp_size = 0;
    float* d_workspace = nullptr;
    hipdnnConvolutionFwdAlgoPerf_t* perf_results = new hipdnnConvolutionFwdAlgoPerf_t[1];

    cudnnGetConvolutionForwardAlgorithm_v7(cudnn, input_desc, filter_desc, conv_desc, output_desc, 1, nullptr,
                                           perf_results);
    falgo = perf_results[0].algo;
    hipdnnGetConvolutionForwardWorkspaceSize(cudnn, input_desc, filter_desc, conv_desc, output_desc, falgo, &temp_size);
    workspace_size = max(workspace_size, temp_size);

    // convolution (bwd - filter)
    hipdnnConvolutionBwdFilterAlgoPerf_t* b_fperf_results = new hipdnnConvolutionBwdFilterAlgoPerf_t[1];
    cudnnGetConvolutionBackwardFilterAlgorithm_v7(cudnn, input_desc, output_desc, conv_desc, filter_desc,
                                                  1, nullptr, b_fperf_results);
    b_falgo = b_fperf_results[0].algo;
    hipdnnGetConvolutionBackwardFilterWorkspaceSize(cudnn, input_desc, output_desc, conv_desc, filter_desc, b_falgo,
                                                   &temp_size);
    workspace_size = max(workspace_size, temp_size);

    // convolution (bwd - data)
    hipdnnConvolutionBwdDataAlgoPerf_t* b_dperf_results = new hipdnnConvolutionBwdDataAlgoPerf_t[1];
    cudnnGetConvolutionBackwardDataAlgorithm_v7(cudnn, filter_desc, output_desc, conv_desc, input_desc,
                                                1, nullptr, b_dperf_results);
    b_dalgo = b_dperf_results[0].algo;
    hipdnnGetConvolutionBackwardDataWorkspaceSize(cudnn, filter_desc, output_desc, conv_desc, input_desc, b_dalgo,
                                                 &temp_size);
    workspace_size = max(workspace_size, temp_size);

    std::cout << "workspace size: " << workspace_size << std::endl;
    std::cout << "[" << __LINE__ << "]" << std::endl;

    // allocate memory space
    hipMalloc((void**) &d_input, sizeof(float) * input_n * input_c * input_h * input_w);
    hipMalloc((void**) &d_filter, sizeof(float) * weight_size);
    hipMalloc((void**) &d_output, sizeof(float) * output_n * output_c * output_h * output_w);
    hipMalloc((void**) &d_workspace, sizeof(float) * workspace_size);
    hipMalloc((void**) &d_bias, sizeof(float) * bias_size);

    hipMemset(d_bias, 0, sizeof(float) * bias_size);

    float* h_input = new float[input_n * input_c * input_h * input_w];
    float* h_filter = new float[weight_size];

    for (int i = 0; i < input_n * input_c * input_h * input_w; i++)
        h_input[i] = i;
    for (int i = 0; i < weight_size; i++)
        h_filter[i] = 0;
    h_filter[1 * 3 + 1] = 1;
    h_filter[3 * 3 + 1 * 3 + 1] = 2;
    hipMemcpy(d_input, h_input, sizeof(float) * input_n * input_c * input_h * input_w, hipMemcpyHostToDevice);
    hipMemcpy(d_filter, h_filter, sizeof(float) * weight_size, hipMemcpyHostToDevice);

    std::cout << "[" << __LINE__ << "]" << std::endl;

    // Forward
    checkCUDNN(hipdnnConvolutionForward(cudnn, &one, input_desc, d_input, filter_desc, d_filter, conv_desc, falgo,
                                       d_workspace, workspace_size, &zero, output_desc, d_output));
    checkCUDNN(hipdnnAddTensor(cudnn, &one, bias_desc, d_bias, &one, output_desc, d_output));
    (hipGetLastError());

    std::cout << "[" << __LINE__ << "]" << std::endl;

    Matrix input(input_h, input_w, input_c, h_input);
    for (int d = 0; d < input_c; d++)
    {
        std::cout << input << std::endl;
        input.GoToNextMatrix();
    }
    input.ResetOffset();
    float* f_output_h = new float[output_n * output_c * output_h * output_w];
    hipMemcpy(f_output_h, d_output, sizeof(float) * output_n * output_c * output_h * output_w,
               hipMemcpyDeviceToHost);
    Matrix output(output_h, output_w, output_c, f_output_h);
    for (int d = 0; d < output_c; d++)
    {
        std::cout << output << std::endl;
        output.GoToNextMatrix();
    }
    output.ResetOffset();

    // backward
    checkCUDNN(hipdnnConvolutionBackwardBias(cudnn, &one, output_desc, d_output, &zero, bias_desc, d_bias));
    checkCUDNN(
            hipdnnConvolutionBackwardFilter(cudnn, &one, input_desc, d_input, output_desc, d_output, conv_desc, b_falgo,
                                           d_workspace, workspace_size, &zero, filter_desc, d_filter));
    checkCUDNN(
            hipdnnConvolutionBackwardData(cudnn, &one, filter_desc, d_filter, output_desc, d_output, conv_desc, b_dalgo,
                                         d_workspace, workspace_size, &zero, input_desc, d_input));
    (hipGetLastError());

    std::cout << "[" << __LINE__ << "]" << std::endl;

    hipdnnDestroyTensorDescriptor(input_desc);
    hipdnnDestroyTensorDescriptor(output_desc);
    hipdnnDestroyFilterDescriptor(filter_desc);
    hipdnnDestroyConvolutionDescriptor(conv_desc);
    hipdnnDestroyTensorDescriptor(bias_desc);

    std::cout << "[" << __LINE__ << "]" << std::endl;

    hipFree(d_input);
    hipFree(d_filter);
    hipFree(d_output);
    hipFree(d_workspace);
    hipFree(d_bias);

    hipdnnDestroy(cudnn);

    std::cout << "[" << __LINE__ << "]" << std::endl;
}*/

/*int main(void)
{
    hipdnnHandle_t cudnn;
    hipdnnCreate(&cudnn);

    Matrix image(5, 5);
    for (int d = 0; d < image.GetDims(); d++)
        for (int i = 0; i < image.GetRows(); i++)
            for (int j = 0; j < image.GetCols(); j++)
                image[d * image.GetRows() * image.GetCols() + i * image.GetCols() + j] = i * image.GetCols() + j;

    const int output_h = image.GetRows() - 2, output_w = image.GetCols() - 2;
    const int numFilters = 2;

    hipdnnTensorDescriptor_t input_descriptor;
    checkCUDNN(hipdnnCreateTensorDescriptor(&input_descriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(input_descriptor,

                                          HIPDNN_TENSOR_NCHW,

                                          HIPDNN_DATA_FLOAT,

                                          1,

                                          image.GetDims(),

                                          image.GetRows(),

                                          image.GetCols()));

    hipdnnTensorDescriptor_t output_descriptor;
    checkCUDNN(hipdnnCreateTensorDescriptor(&output_descriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(output_descriptor,

                                          HIPDNN_TENSOR_NCHW,

                                          HIPDNN_DATA_FLOAT,

                                          1,

                                          numFilters,

                                          output_h,

                                          output_w));

    hipdnnFilterDescriptor_t kernel_descriptor;
    checkCUDNN(hipdnnCreateFilterDescriptor(&kernel_descriptor));
    checkCUDNN(hipdnnSetFilter4dDescriptor(kernel_descriptor,

                                          HIPDNN_DATA_FLOAT,

                                          HIPDNN_TENSOR_NCHW,

                                          numFilters,

                                          image.GetDims(),
                                          3,
                                          3));

    hipdnnConvolutionDescriptor_t convolution_descriptor;
    checkCUDNN(hipdnnCreateConvolutionDescriptor(&convolution_descriptor));
    checkCUDNN(hipdnnSetConvolution2dDescriptor(convolution_descriptor,

                                               0,

                                               0,

                                               1,

                                               1,

                                               1,

                                               1,

                                               HIPDNN_CROSS_CORRELATION,

                                               HIPDNN_DATA_FLOAT));

    hipdnnConvolutionFwdAlgo_t convolution_algorithm;
    const int requested_algo_count = 1;
    int returned_algo_count;
    hipdnnConvolutionFwdAlgoPerf_t* convolution_algorithms = new hipdnnConvolutionFwdAlgoPerf_t[requested_algo_count];
    checkCUDNN(
            cudnnGetConvolutionForwardAlgorithm_v7(cudnn,
                                                   input_descriptor,
                                                   kernel_descriptor,
                                                   convolution_descriptor,
                                                   output_descriptor,
                                                   requested_algo_count,
                                                   &returned_algo_count,
                                                   convolution_algorithms));
    convolution_algorithm = convolution_algorithms[0].algo;

    size_t workspace_bytes = 0;
    checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(cudnn,
                                                       input_descriptor,
                                                       kernel_descriptor,
                                                       convolution_descriptor,
                                                       output_descriptor,
                                                       convolution_algorithm,
                                                       &workspace_bytes));
    std::cerr << "Workspace size: " << (workspace_bytes / 1048576.0) << "MB"
              << std::endl;

    int o_batch_size{0}, o_channels{0}, o_height{0}, o_width{0};
    checkCUDNN(hipdnnGetConvolution2dForwardOutputDim(convolution_descriptor,
                                                     input_descriptor,
                                                     kernel_descriptor,
                                                     &o_batch_size,
                                                     &o_channels,
                                                     &o_height,
                                                     &o_width));
    if (numFilters != o_channels)
    {
        std::cerr << "Input/output o_channels do not match network configuration."
                  << std::endl;
        return 1;
    }
    if (output_h != o_height || output_w != o_width)
    {
        std::cerr << "Input/output dimensions do not match network configuration."
                  << std::endl;
        return 1;
    }
    if (o_batch_size != 1)
    {
        std::cerr << "Invalid batch size." << std::endl;
        return 1;
    }

    void* d_workspace{nullptr};
    if (workspace_bytes)
        hipMalloc(&d_workspace, workspace_bytes);

    const int image_bytes = o_batch_size * image.GetDims() * image.GetRows() * image.GetCols() * sizeof(float);
    const int output_bytes = o_batch_size * numFilters * output_h * output_w * sizeof(float);

    float* d_input{nullptr};
    hipMalloc(&d_input, image_bytes);
    hipMemcpy(d_input, image.GetData(), image_bytes, hipMemcpyHostToDevice);

    float* d_output{nullptr};
    hipMalloc(&d_output, output_bytes);
    hipMemset(d_output, 0, output_bytes);


    const int kernelSize = numFilters * image.GetDims() * 3 * 3;
    float* h_filter = new float[kernelSize];
    for (int i = 0; i < kernelSize; i++)
        h_filter[i] = 0;
    h_filter[1 * 3 + 1] = 1;
    h_filter[3 * 3 + 1 * 3 + 1] = 2;


    float* d_kernel{nullptr};
    hipMalloc(&d_kernel, kernelSize * sizeof(float));
    hipMemcpy(d_kernel, h_filter, kernelSize * sizeof(float), hipMemcpyHostToDevice);

    const float alpha = 1, beta = 0;
    checkCUDNN(hipdnnConvolutionForward(cudnn,
                                       &alpha,
                                       input_descriptor,
                                       d_input,
                                       kernel_descriptor,
                                       d_kernel,
                                       convolution_descriptor,
                                       convolution_algorithm,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       output_descriptor,
                                       d_output));

    float* h_output = new float[output_bytes / sizeof(float)];
    hipMemcpy(h_output, d_output, output_bytes, hipMemcpyDeviceToHost);

    for (int i = 0; i < image.GetDims(); i++)
    {
        std::cout << image << std::endl;
        image.GoToNextMatrix();
    }
    image.ResetOffset();
    CloneMatrix res(output_h, output_w, numFilters, h_output);
    for (int i = 0; i < numFilters; i++)
    {
        std::cout << res << std::endl;
        res.GoToNextMatrix();
    }
    res.ResetOffset();

    //delete[] h_output;
    hipFree(d_kernel);
    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_workspace);

    hipdnnDestroyTensorDescriptor(input_descriptor);
    hipdnnDestroyTensorDescriptor(output_descriptor);
    hipdnnDestroyFilterDescriptor(kernel_descriptor);
    hipdnnDestroyConvolutionDescriptor(convolution_descriptor);

    hipdnnDestroy(cudnn);

    return 0;;
}*/
