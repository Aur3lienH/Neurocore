#include "Examples/Mnist.cuh"

#define TEST_ACTIVATION_BACKWARD 0

int main()
{
#if TEST_ACTIVATION_BACKWARD

    float alpha = 1.0f;
    float beta = 0.0f;
    hipdnnCreate(&cudnnHandle);
    hipblasCreate(&cublasHandle);

    hipdnnActivationDescriptor_t activationDesc;
    checkCUDNN(hipdnnCreateActivationDescriptor(&activationDesc));
    checkCUDNN(hipdnnSetActivationDescriptor(activationDesc, HIPDNN_ACTIVATION_RELU, HIPDNN_NOT_PROPAGATE_NAN, 0));

    Matrix values(28, 28);
    for (int i = 0; i < 28 * 28; i++)
        values[i] = i - 5;

    float* forward_input_d;
    checkCUDA(hipMalloc(&forward_input_d, 28 * 28 * sizeof(float)));
    checkCUDA(hipMemcpy(forward_input_d, values.GetData(), 28 * 28 * sizeof(float), hipMemcpyHostToDevice));
    hipdnnTensorDescriptor_t forwardInputDescriptor;
    checkCUDNN(hipdnnCreateTensorDescriptor(&forwardInputDescriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(forwardInputDescriptor,
                                          HIPDNN_TENSOR_NCHW,
                                          HIPDNN_DATA_FLOAT,
                                          1,
                                          1,
                                          28,
                                          28));

    float* forward_output_d;
    checkCUDA(hipMalloc(&forward_output_d, 28 * 28 * sizeof(float)));
    hipdnnTensorDescriptor_t forwardOutputDescriptor;
    checkCUDNN(hipdnnCreateTensorDescriptor(&forwardOutputDescriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(forwardOutputDescriptor,
                                          HIPDNN_TENSOR_NCHW,
                                          HIPDNN_DATA_FLOAT,
                                          1,
                                          1,
                                          28,
                                          28));

    checkCUDNN(hipdnnActivationForward(cudnnHandle,
                                      activationDesc,
                                      &alpha,
                                      forwardInputDescriptor,
                                      forward_input_d,
                                      &beta,
                                      forwardOutputDescriptor,
                                      forward_output_d));

    float* forward_output = new float[28 * 28];
    checkCUDA(hipMemcpy(forward_output, forward_output_d, 28 * 28 * sizeof(float), hipMemcpyDeviceToHost));

    std::cout << "Values: \n";
    for (int i = 0; i < 28 * 28; i++)
        std::cout << values[i] << " ";

    std::cout << std::endl;

    std::cout << "Forward output: \n";
    for (int i = 0; i < 28 * 28; i++)
        std::cout << forward_output[i] << " ";

    float* ones_d;
    checkCUDA(hipMalloc(&ones_d, 28 * 28 * sizeof(float)));
    checkCUDA(hipMemset(ones_d, 1, 28 * 28 * sizeof(float)));
    hipdnnTensorDescriptor_t onesDesc;
    checkCUDNN(hipdnnCreateTensorDescriptor(&onesDesc));
    checkCUDNN(hipdnnSetTensor4dDescriptor(onesDesc,
                                          HIPDNN_TENSOR_NCHW,
                                          HIPDNN_DATA_FLOAT,
                                          1,
                                          1,
                                          28,
                                          28));

    float* backward_output_d;
    checkCUDA(hipMalloc(&backward_output_d, 28 * 28 * sizeof(float)));
    hipdnnTensorDescriptor_t backward_output_desc;
    checkCUDNN(hipdnnCreateTensorDescriptor(&backward_output_desc));
    checkCUDNN(hipdnnSetTensor4dDescriptor(backward_output_desc,
                                          HIPDNN_TENSOR_NCHW,
                                          HIPDNN_DATA_FLOAT,
                                          1,
                                          1,
                                          28,
                                          28));

    float* backward_delta_output_d;
    checkCUDA(hipMalloc(&backward_delta_output_d, 28 * 28 * sizeof(float)));
    hipdnnTensorDescriptor_t backward_delta_output_desc;
    checkCUDNN(hipdnnCreateTensorDescriptor(&backward_delta_output_desc));
    checkCUDNN(hipdnnSetTensor4dDescriptor(backward_delta_output_desc,
                                          HIPDNN_TENSOR_NCHW,
                                          HIPDNN_DATA_FLOAT,
                                          1,
                                          1,
                                          28,
                                          28));

    checkCUDNN(hipdnnActivationBackward(cudnnHandle,
                                       activationDesc,
                                       &alpha,
                                       forwardOutputDescriptor,
                                       forward_output_d,
                                       onesDesc,
                                       ones_d,
                                       backward_output_desc,
                                       backward_output_d,
                                       &beta, backward_delta_output_desc,
                                       backward_delta_output_d));

    float* backward_output = new float[28 * 28];
    checkCUDA(hipMemcpy(backward_output, backward_output_d, 28 * 28 * sizeof(float), hipMemcpyDeviceToHost));

    float* backward_delta_output = new float[28 * 28];
    checkCUDA(hipMemcpy(backward_delta_output, backward_delta_output_d, 28 * 28 * sizeof(float),
                         hipMemcpyDeviceToHost));

    std::cout << std::endl;

    std::cout << "Backward output: \n";
    for (int i = 0; i < 28 * 28; i++)
        std::cout << backward_output[i] << " ";

    std::cout << std::endl;

    std::cout << "Backward delta output: \n";
    for (int i = 0; i < 28 * 28; i++)
        std::cout << backward_delta_output[i] << " ";

    return 0;
#endif
    Mnist1();
    //QuickDraw2(10000);

#if USE_GPU
    delete Matrix_GPU::cuda;
#endif
    return 0;
    //LoadAndTest("./Models/MNIST_11.net",true);
}